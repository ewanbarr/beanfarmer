#include "hip/hip_runtime.h"
/*
Copyright (c) 2017 Ewan D. Barr

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.

Maintainer: Ewan D. Barr (ebarr@mpifr-bonn.mpg.de)
*/
#include "params.h"
#include "cuda_tools.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_complex.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <vector>
#include <random>
#include <sstream>
#include <stdexcept>

struct ComplexInt8
{
  int8_t x;
  int8_t y;
};

/**
 * @brief      Perform beamforming followed by detection and integration in time.
 *
 * @param      aptf_voltages  Raw voltages in antenna, polarisation, time, frequency order (fastest to slowest)
 * @param      tbf_powers     Output detected integrated powers in time, frequency order (fastest to slowest)
 */
__global__
void icbf_aptf_general_k
(
 char2 const* __restrict__ aptf_voltages,
 float* __restrict__ tf_powers,
 float const* __restrict__ weights)
{
  /**
   * Each warp reads all the data it requires and performs detection followed
   * by a warp reduce. The resultant sums are transposed and the first warp of
   * each block is left to write all results back to global memory.
   */
  static_assert(NSAMPLES%NSAMPLES_PER_BLOCK==0,
		"Kernel can only process a multiple of (NWARPS_PER_BLOCK * NACCUMULATE) samples.");
  static_assert(NTHREADS%WARP_SIZE==0,
		"Number of threads must be an integer multiple of WARP_SIZE.");

  volatile __shared__ float temp[WARP_SIZE][WARP_SIZE];
  volatile __shared__ float shared_weights[NANTENNAS];
  int const warp_idx = threadIdx.x / 0x20;
  int const lane_idx = threadIdx.x & 0x1f;
  int sample_offset = NACCUMULATE * (blockIdx.x * NWARPS_PER_BLOCK + warp_idx);
  int aptf_voltages_partial_idx = NANTENNAS * NPOL * (NSAMPLES * blockIdx.y + sample_offset);


  for (int antenna_idx = threadIdx.x; antenna_idx < NANTENNAS; antenna_idx += blockDim.x)
  {
    shared_weights[antenna_idx] = weights[antenna_idx];
  }


  //Accumulators for 8-bit complex detection and additions
  int xx = 0, yy = 0;

  for (int offset = lane_idx; offset < NANTENNAS*NPOL*NACCUMULATE; offset += WARP_SIZE)
    {
      int antenna_idx = offset % NANTENNAS;
      float weight = shared_weights[antenna_idx];
      char2 ant = aptf_voltages[aptf_voltages_partial_idx  + offset];
      xx += (ant.x * ant.x) * weight;
      yy += (ant.y * ant.y) * weight;
    }
  //Form power and write to shared memory
  temp[warp_idx][lane_idx] = (float)(xx + yy);
  __syncthreads();

  //Warp reduce
  if (lane_idx < 16)
    {
      for (int src_lane = 16; src_lane > 0; src_lane >>= 1)
	temp[warp_idx][lane_idx] += temp[warp_idx][lane_idx+src_lane];
    }
  __syncthreads();

  //Transpose shared memory
  if (lane_idx==0)
      temp[0][warp_idx] = temp[warp_idx][0];
  __syncthreads();

  //First warp writes back to global memory
  if (warp_idx==0)
    {
      int output_idx = (NWARPS_PER_BLOCK * gridDim.x) * blockIdx.y
	+ (blockIdx.x * NWARPS_PER_BLOCK + lane_idx);
      tf_powers[output_idx] = temp[0][lane_idx];
    }
}

void icbf_reference_cpp
(
 ComplexInt8 const* __restrict__ aptf_voltages,
 float* __restrict__ tf_powers,
 float const* __restrict__ weights)
{
  for (int channel_idx = 0; channel_idx < NCHANNELS; ++channel_idx)
    {
      for (int sample_idx = 0; sample_idx < NSAMPLES; sample_idx+=NACCUMULATE)
	{
	  float power = 0.0f;
	  for (int sample_offset = 0; sample_offset < NACCUMULATE; ++sample_offset)
	    {
	      for (int pol_idx = 0; pol_idx < NPOL; ++pol_idx)
		{
		  hipComplex accumulator = make_hipComplex(0.0f,0.0f);
		  for (int antenna_idx = 0; antenna_idx < NANTENNAS; ++antenna_idx)
		    {
		      int aptf_voltages_idx = NANTENNAS * NPOL * NSAMPLES * channel_idx
			+ NANTENNAS * NPOL * (sample_idx + sample_offset)
			+ NANTENNAS * pol_idx
			+ antenna_idx;
		      ComplexInt8 ant = aptf_voltages[aptf_voltages_idx];
		      power += weights[antenna_idx] * (ant.x*ant.x + ant.y*ant.y);
		    }
		}
	    }
	  int tf_powers_idx = NSAMPLES/NACCUMULATE  * channel_idx
	    + sample_idx/NACCUMULATE;
	  tf_powers[tf_powers_idx] = power;
	}
    }
}


bool is_same(float* a, float*b, std::size_t size, float tolerance)
{
  for (std::size_t idx = 0; idx < size; ++idx)
    {
      if (abs((a[idx]-b[idx])/a[idx]) >= tolerance)
	{
	  std::cout << "Expected " << a[idx] << " got " << b[idx] << "\n";
	  return false;
	}
    }
  return true;
}

template <typename ComplexType>
void populate(ComplexType* data, std::size_t size, int lower, int upper)
{
  std::random_device rd;
  std::mt19937 eng(rd());
  std::uniform_int_distribution<> distr(lower, upper);
  for(std::size_t n = 0 ; n < size; ++n)
    {
      data[n].x = distr(eng);
      data[n].y = distr(eng);
    }
}

int main()
{
  std::size_t aptf_voltages_size = NPOL * NSAMPLES * NANTENNAS * NCHANNELS;
  std::size_t tbf_powers_size = NSAMPLES/NACCUMULATE * NCHANNELS;
  std::cout << "PTA array size: " << aptf_voltages_size << "\n";
  std::cout << "output size: " << tbf_powers_size << "\n";
  std::cout << "Global memory required: "
	    << (tbf_powers_size * sizeof(float)
		+ aptf_voltages_size*sizeof(ComplexInt8))/1.0e9
	    << "GB \n";

  /**
   * Currently we are only considering 4k mode on the channeliser
   */;
  float duration = TSAMP * NSAMPLES;
  std::cout << "Duration of data: " << duration << " seconds" << std::endl;

  CUDA_ERROR_CHECK(hipSetDevice(0));
  CUDA_ERROR_CHECK(hipDeviceReset());

  /**
   * Below we set default values for the arrays. Beamforming this data should result in
   * every output having the same value.
   *
   */
#ifdef TEST_CORRECTNESS
  std::cout << "Generating host test vectors...\n";
  ComplexInt8 default_value = {0,0};
  thrust::host_vector<ComplexInt8> pta_vector_h(aptf_voltages_size,default_value);
  thrust::host_vector<float> weights_vector_h(NANTENNAS, 1.0f);
  populate<ComplexInt8>(pta_vector_h.data(),aptf_voltages_size,-10,10);
  weights_vector_h[NANTENNAS/2] = 0.0f;
  weights_vector_h[NANTENNAS/4] = 0.0f;
  thrust::device_vector<ComplexInt8> pta_vector = pta_vector_h;
  thrust::device_vector<float> weights_vector = weights_vector_h;
#else
  std::cout << "NOT generating host test vectors...\n";
  thrust::device_vector<ComplexInt8> pta_vector(aptf_voltages_size);
  thrust::device_vector<float> weights_vector(NANTENNAS, 1.0f);
#endif //TEST_CORRECTNESS

  thrust::device_vector<float> output_vector(tbf_powers_size,0.0f);
  ComplexInt8 const* aptf_voltages = thrust::raw_pointer_cast(pta_vector.data());
  float const* weights_ptr = thrust::raw_pointer_cast(weights_vector.data());
  float* tbf_powers = thrust::raw_pointer_cast(output_vector.data());
  dim3 grid(NSAMPLES/(NWARPS_PER_BLOCK*NACCUMULATE), NCHANNELS);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  std::cout << "Executing warm up\n";
  //Warm up
  for (int jj=0; jj<NITERATIONS; ++jj)
    icbf_aptf_general_k<<<grid,NTHREADS>>>((char2*)aptf_voltages, tbf_powers, weights_ptr);
  CUDA_ERROR_CHECK(hipDeviceSynchronize());

  std::cout << "Starting benchmarking\n";
  hipEventRecord(start);
  for (int ii=0; ii<NITERATIONS; ++ii)
    icbf_aptf_general_k<<<grid,NTHREADS>>>((char2*)aptf_voltages, tbf_powers, weights_ptr);
  CUDA_ERROR_CHECK(hipDeviceSynchronize());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "Total kernel duration (ms): " << milliseconds << "\n";

#ifdef TEST_CORRECTNESS
  std::cout << "Testing correctness...\n";
  thrust::host_vector<float> gpu_output = output_vector;
  thrust::host_vector<float> cpu_output(tbf_powers_size);
  CUDA_ERROR_CHECK(hipDeviceSynchronize());
  icbf_reference_cpp(pta_vector_h.data(), cpu_output.data(), weights_vector_h.data());
  if (!is_same(cpu_output.data(),gpu_output.data(), NSAMPLES/NACCUMULATE*NCHANNELS, 0.001))
    std::cout << "FAILED!\n";
  else
    std::cout << "PASSED!\n";

#endif

  return 0;
}
